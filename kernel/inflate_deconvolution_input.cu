#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>

#include "convolution_param.h"

__global__ void Copy(
    float* dst, const float* src, int total_size,
    struct ConvolutionParam param) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id >= total_size) {
        return;
    }

    int input_h = param.mH;
    int input_w = param.mW;
    int channel = global_id / input_h / input_w;
    int x = global_id % (input_h * input_w) / input_w;
    int y = global_id % (input_h * input_w) % input_w;

    int output_h = (param.mH - 1) * param.mStrideH + 1;
    int output_w = (param.mW - 1) * param.mStrideW + 1;
    int output_x = x * param.mStrideH;
    int output_y = y * param.mStrideW;

    dst[channel * output_h * output_w + output_x * output_w + output_y] =
        src[global_id];
}

float* InflateDeconvolutionInput(
    const float* src, struct ConvolutionParam param, hipStream_t stream) {
    float* dst;

    int total_size = param.mInputChannel * param.mH * param.mW;

    int output_h = (param.mH - 1) * param.mStrideH + 1;
    int output_w = (param.mW - 1) * param.mStrideW + 1;

    hipMallocManaged(&dst, param.mInputChannel * output_h * output_w * 4);
    hipMemset(dst, 0, param.mInputChannel * output_h * output_w * 4);
    Copy<<<int(total_size / 128) + 1, 128, 0, stream>>>(
        dst, src, total_size, param);

    return dst;
}
